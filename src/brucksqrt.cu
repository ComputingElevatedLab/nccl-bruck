#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2016-2022, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/
#include <chrono>
#include <cstring>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <vector>

#include "common.h"
#include "hip/hip_runtime.h"

void BrucksqrtGetCollByteCount(size_t *sendcount, size_t *recvcount, size_t *paramcount, size_t *sendInplaceOffset, size_t *recvInplaceOffset, size_t count, int nranks)
{
    *sendcount = (count / nranks) * nranks;
    *recvcount = (count / nranks) * nranks;
    *sendInplaceOffset = 0;
    *recvInplaceOffset = 0;
    *paramcount = count / nranks;
}

testResult_t BrucksqrtInitData(struct threadArgs *args, ncclDataType_t type, ncclRedOp_t op, int root, int rep, int in_place)
{
    size_t sendcount = args->sendBytes / wordSize(type);
    size_t recvcount = args->expectedBytes / wordSize(type);
    int nranks = args->nProcs * args->nThreads * args->nGpus;

    for (int i = 0; i < args->nGpus; i++)
    {
        CUDACHECK(hipSetDevice(args->gpus[i]));
        int rank = ((args->proc * args->nThreads + args->thread) * args->nGpus + i);
        CUDACHECK(hipMemset(args->recvbuffs[i], 0, args->expectedBytes));
        void *data = in_place ? args->recvbuffs[i] : args->sendbuffs[i];
        TESTCHECK(InitData(data, sendcount, 0, type, ncclSum, 33 * rep + rank, 1, 0));
        for (int j = 0; j < nranks; j++)
        {
            size_t partcount = sendcount / nranks;
            TESTCHECK(InitData((char *)args->expected[i] + j * partcount * wordSize(type), partcount, rank * partcount, type, ncclSum, 33 * rep + j, 1, 0));
        }
        CUDACHECK(hipDeviceSynchronize());
    }
    // We don't support in-place brucksqrt
    args->reportErrors = in_place ? 0 : 1;
    return testSuccess;
}

void BrucksqrtGetBw(size_t count, int typesize, double sec, double *algBw, double *busBw, int nranks)
{
    double baseBw = (double)(count * nranks * typesize) / 1.0E9 / sec;

    *algBw = baseBw;
    double factor = ((double)(nranks - 1)) / ((double)(nranks));
    *busBw = baseBw * factor;
}

int myPow(int x, unsigned int p)
{
    if (p == 0)
    {
        return 1;
    }
    else if (p == 1)
    {
        return x;
    }

    int tmp = myPow(x, p / 2);
    if (p % 2 == 0)
    {
        return tmp * tmp;
    }
    else
    {
        return x * tmp * tmp;
    }
}

std::vector<int> convert10tob(int w, int N, int b)
{
    std::vector<int> v(w);
    int i = 0;
    while (N)
    {
        v[i++] = (N % b);
        N /= b;
    }
    return v;
}

testResult_t BrucksqrtRunColl(void *sendbuff, void *recvbuff, size_t count, ncclDataType_t type, ncclRedOp_t op, int root, ncclComm_t comm, hipStream_t stream)
{
#if NCCL_MAJOR < 2 || NCCL_MINOR < 7
    printf("NCCL 2.7 or later is needed for brucksqrt. This test was compiled with %d.%d.\n", NCCL_MAJOR, NCCL_MINOR);
    return testNcclError;
#else
    int nprocs, rank;
    NCCLCHECK(ncclCommCount(comm, &nprocs));
    NCCLCHECK(ncclCommUserRank(comm, &rank));
    size_t unit_size = count * wordSize(type);

    char *c_sendbuff = (char *)sendbuff;
    char *c_recvbuff = (char *)recvbuff;
    int radix = std::ceil(std::sqrt(nprocs));
    int w = std::ceil(std::log(nprocs) / std::log(radix));

    int pows[w];
    for (int i = 0; i <= w; i++)
    {
        pows[i] = myPow(radix, i);
    }

    int nlpow = pows[w - 1];
    int d = (pows[w] - nprocs) / nlpow;

    CUDACHECK(hipMemcpyAsync(c_recvbuff, c_sendbuff, nprocs * count, hipMemcpyDeviceToDevice, stream));
    CUDACHECK(hipMemcpyAsync(&c_sendbuff[(nprocs - rank) * count], c_recvbuff, rank * count, hipMemcpyDeviceToDevice, stream));
    CUDACHECK(hipMemcpyAsync(c_sendbuff, &c_recvbuff[rank * count], (nprocs - rank) * count, hipMemcpyDeviceToDevice, stream));

    int *rank_r_reps = new int[nprocs * w * sizeof(int)];
    for (int i = 0; i < nprocs; i++)
    {
        std::vector<int> r_rep = convert10tob(w, i, radix);
        std::memcpy(&rank_r_reps[i * w], r_rep.data(), w * sizeof(int));
    }

    int sent_blocks[nlpow];
    int di = 0;
    int ci = 0;

    char *tempbuff;
    CUDACHECK(hipMallocAsync((void **)&tempbuff, nlpow * count, stream));
    for (int x = 0; x < w; x++)
    {
        int ze = (x == w - 1) ? radix - d : radix;
        for (int z = 1; z < ze; z++)
        {
            di = 0;
            ci = 0;
            for (int i = 0; i < nprocs; i++)
            {
                if (rank_r_reps[i * w + x] == z)
                {
                    sent_blocks[di++] = i;
                    CUDACHECK(hipMemcpyAsync(&tempbuff[count * ci++], &c_sendbuff[count * i], count, hipMemcpyDeviceToDevice, stream));
                }
            }

            int distance = z * pows[x];
            int recv_proc = (rank - distance + nprocs) % nprocs;
            int send_proc = (rank + distance) % nprocs;
            long long comm_size = di * count;
            NCCLCHECK(ncclGroupStart());
            NCCLCHECK(ncclSend(tempbuff, comm_size, ncclChar, send_proc, comm, stream));
            NCCLCHECK(ncclRecv(c_recvbuff, comm_size, ncclChar, recv_proc, comm, stream));
            NCCLCHECK(ncclGroupEnd());

            for (int i = 0; i < di; i++)
            {
                long long offset = sent_blocks[i] * count;
                CUDACHECK(hipMemcpyAsync(c_sendbuff + offset, c_recvbuff + (count * i), count, hipMemcpyDeviceToDevice, stream));
            }
        }
    }

    for (int i = 0; i < nprocs; i++)
    {
        int index = (rank - i + nprocs) % nprocs;
        CUDACHECK(hipMemcpyAsync(&c_recvbuff[count * index], &c_sendbuff[count * i], count, hipMemcpyDeviceToDevice, stream));
    }

    CUDACHECK(hipFreeAsync(tempbuff, stream));

    return testSuccess;
#endif
}

struct testColl brucksqrtTest = {
    "Brucksqrt",
    BrucksqrtGetCollByteCount,
    BrucksqrtInitData,
    BrucksqrtGetBw,
    BrucksqrtRunColl};

void BrucksqrtGetBuffSize(size_t *sendcount, size_t *recvcount, size_t count, int nranks)
{
    size_t paramcount, sendInplaceOffset, recvInplaceOffset;
    BrucksqrtGetCollByteCount(sendcount, recvcount, &paramcount, &sendInplaceOffset, &recvInplaceOffset, count, nranks);
}

testResult_t BrucksqrtRunTest(struct threadArgs *args, int root, ncclDataType_t type, const char *typeName, ncclRedOp_t op, const char *opName)
{
    args->collTest = &brucksqrtTest;
    ncclDataType_t *run_types;
    const char **run_typenames;
    int type_count;

    if ((int)type != -1)
    {
        type_count = 1;
        run_types = &type;
        run_typenames = &typeName;
    }
    else
    {
        type_count = test_typenum;
        run_types = test_types;
        run_typenames = test_typenames;
    }

    for (int i = 0; i < type_count; i++)
    {
        TESTCHECK(TimeTest(args, run_types[i], run_typenames[i], (ncclRedOp_t)0, "none", -1));
    }
    return testSuccess;
}

struct testEngine brucksqrtEngine = {
    BrucksqrtGetBuffSize,
    BrucksqrtRunTest};

#pragma weak ncclTestEngine = brucksqrtEngine
